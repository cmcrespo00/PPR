#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string.h>
#include <time.h>
#include "Graph.h"

using namespace std;

//#define blocksize 64
//#define blocksize_ 8
//#define blocksize 256
//#define blocksize_ 16
#define blocksize 1024
#define blocksize_ 32

//**************************************************************************
//*****************************KERNEL ORIGINAL******************************
__global__ void floyd_kernel_original(int * M, const int nverts, const int k) {
  int ij = threadIdx.x + blockDim.x * blockIdx.x;
  int i= ij / nverts;
  int j= ij - i * nverts;
  if (i<nverts && j< nverts) {
    int Mij = M[ij];
    if (i != j && i != k && j != k) {
      int Mikj = M[i * nverts + k] + M[k * nverts + j];
      Mij = (Mij > Mikj) ? Mikj : Mij;
      M[ij] = Mij;
    }
  }
}
//**************************************************************************

//**************************************************************************
//****************************KERNEL MODIFICADO*****************************
__global__ void floyd_kernel(int * M, const int nverts, const int k) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;  // Compute row index
  int i = blockIdx.y * blockDim.y + threadIdx.y;  // Compute column index

  int ij=i*nverts+j; // Compute global 1D index

  if (i < nverts && j < nverts) {
    int Mij = M[ij];
    if (i != j && i != k && j != k) {
      int Mikj = M[i * nverts + k] + M[k * nverts + j];
      Mij = (Mij > Mikj) ? Mikj : Mij;
      M[ij] = Mij;
    }
  }
}
//**************************************************************************

//**************************************************************************
//**************************** KERNEL REDUCCION ****************************
__global__ void floyd_reduccion(int *M, long int *suma, const int nverts2) {
	extern __shared__ float sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = ((i < nverts2) ? M[i] : 0.0f);
	__syncthreads();

	for(int s = blockDim.x/2; s > 0; s >>= 1){
  	if (tid < s)
  	  sdata[tid] += sdata[tid+s];
  	__syncthreads();
	}
	if (tid == 0)
    suma[blockIdx.x] = sdata[0];
}
//**************************************************************************

//**************************************************************************
// ************  MAIN FUNCTION *********************************************
int main (int argc, char *argv[]) {
  double time, Tcpu, Tgpu_original, Tgpu_modificado,
               Tcpu_reduccion, Tgpu_reduccion;

  if (argc != 2) {
    cerr << "Sintaxis: " << argv[0] << " <archivo de grafo>" << endl;
    return(-1);
  }

  //Get GPU information
  int num_devices,devID;
  hipDeviceProp_t props;
  hipError_t err;

  err=hipGetDeviceCount(&num_devices);
  if (err == hipSuccess) {
    cout <<endl<< num_devices <<" CUDA-enabled  GPUs detected in this computer system"<<endl<<endl;
    cout<<"....................................................."<<endl<<endl;
  }
  else {
    cerr << "ERROR detecting CUDA devices......" << endl; exit(-1);
  }

  for (int i = 0; i < num_devices; i++) {
    devID=i;
    err = hipGetDeviceProperties(&props, devID);
    cout<<"Device "<<devID<<": "<< props.name <<" with Compute Capability: "<<props.major<<"."<<props.minor<<endl<<endl;

    if (err != hipSuccess) {
      cerr << "ERROR getting CUDA devices" << endl;
    }
  }

  devID = 0;
  cout<<"Using Device "<<devID<<endl;
  cout<<"....................................................."<<endl<<endl;

  err = hipSetDevice(devID);
  if(err != hipSuccess) {
    cerr << "ERROR setting CUDA device" <<devID<< endl;
  }

  // Declaration of the Graph object
  Graph G;

  // Read the Graph
  G.lee(argv[1]);

  //cout << "The input Graph:"<<endl;
  //G.imprime();
  const int nverts = G.vertices;
  const int niters = nverts;
  const int nverts2 = nverts * nverts;

  int *c_Out_M = new int[nverts2];
  int size = nverts2*sizeof(int);
  int * d_In_M = NULL;

  err = hipMalloc((void **) &d_In_M, size);
  if (err != hipSuccess) {
    cerr << "ERROR MALLOC" << endl;
  }

  // Get the integer 2D array for the dense graph
  int *A = G.Get_Matrix();

  //**************************************************************************
  // GPU phase ORIGINAL
  //**************************************************************************

  time=clock();

  err = hipMemcpy(d_In_M, A, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR CUDA MEM. COPY" << endl;
  }

  // Main Loop
  for(int k = 0; k < niters; k++) {
    //printf("CUDA kernel launch \n");
    int threadsPerBlock_original = blocksize;
    int blocksPerGrid_original = (nverts2 + threadsPerBlock_original - 1) / threadsPerBlock_original;
    // Kernel Launch
    floyd_kernel_original<<<blocksPerGrid_original,threadsPerBlock_original>>>(d_In_M, nverts, k);
    err = hipGetLastError();

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel! ERROR= %d\n",err);
      exit(EXIT_FAILURE);
    }
  }
  err =hipMemcpy(c_Out_M, d_In_M, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "ERROR CUDA MEM. COPY" << endl;
  }

  Tgpu_original=(clock()-time)/CLOCKS_PER_SEC;

  cout << "Time spent on GPU original version = " << Tgpu_original << endl << endl;


  //**************************************************************************
  // GPU phase MODIFICADO
  //**************************************************************************

  time=clock();

  err = hipMemcpy(d_In_M, A, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "ERROR CUDA MEM. COPY" << endl;
  }

  // Main Loop
  for(int k = 0; k < niters; k++) {
    //printf("CUDA kernel launch \n");
    dim3 threadsPerBlock_modificado (blocksize_, blocksize_);
    dim3 blocksPerGrid_modificado( ceil ((float)(nverts)/threadsPerBlock_modificado.x), ceil ((float)(nverts)/threadsPerBlock_modificado.y) );
    // Kernel Launch
    floyd_kernel<<<blocksPerGrid_modificado,threadsPerBlock_modificado>>>(d_In_M, nverts, k);
    err = hipGetLastError();

    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel! ERROR= %d\n",err);
      exit(EXIT_FAILURE);
    }
  }
  err =hipMemcpy(c_Out_M, d_In_M, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "ERROR CUDA MEM. COPY" << endl;
  }

  Tgpu_modificado=(clock()-time)/CLOCKS_PER_SEC;

  cout << "Time spent on GPU modified version= " << Tgpu_modificado << endl << endl;

  //**************************************************************************
  //***************************** REDUCCION GPU ******************************
  //**************************************************************************
  dim3 threadsPerBlock_reduccion(blocksize, 1);
  dim3 numBlocks_reduccion( ceil ((float)(nverts2)/threadsPerBlock_reduccion.x), 1);

  // Vector suma en CPU
  long int *suma_GPU;
  suma_GPU = (long int*) malloc(numBlocks_reduccion.x*sizeof(long int));

  // Minimum vector  to be computed on GPU
  long int *suma_GPU_d;
  hipMalloc ((void **) &suma_GPU_d, sizeof(float)*numBlocks_reduccion.x);

  int smemSize_reduccion = threadsPerBlock_reduccion.x*sizeof(float);

  time = clock();
  // Llamada al kernel de reduccion
  floyd_reduccion<<<numBlocks_reduccion, threadsPerBlock_reduccion, smemSize_reduccion>>>(d_In_M, suma_GPU_d, nverts2);

  Tgpu_reduccion = (clock() - time) / CLOCKS_PER_SEC;

  /* Copy data from device memory to host memory */
  hipMemcpy(suma_GPU, suma_GPU_d, numBlocks_reduccion.x*sizeof(float),hipMemcpyDeviceToHost);

  // Perform final reduction in CPU
  long int sumaGPU = 0;
  float mediaGPU;
  for (int i = 0; i < numBlocks_reduccion.x; i++)
    sumaGPU += suma_GPU[i];
  mediaGPU = sumaGPU / nverts2;

  cout << "Tiempo GPU algoritmo reduccion = " << Tgpu_reduccion << endl;
  cout << "Suma obtenida en el algoritmo de reduccion GPU = " << sumaGPU << endl;
  cout << "Media obtenida en el algoritmo de reduccion GPU = " << mediaGPU << endl << endl;
  cout<<"....................................................."<<endl<<endl;

  //**************************************************************************
  // CPU phase
  //**************************************************************************

  time=clock();

  // BUCLE PPAL DEL ALGORITMO
  int inj, in, kn;
  for(int k = 0; k < niters; k++) {
    kn = k * nverts;
    for(int i=0;i<nverts;i++) {
      in = i * nverts;
      for(int j = 0; j < nverts; j++)
        if (i!=j && i!=k && j!=k){
          inj = in + j;
          A[inj] = min(A[in+k] + A[kn+j], A[inj]);
        }
    }
  }

  Tcpu=(clock()-time)/CLOCKS_PER_SEC;
  cout << "Time spent on CPU= " << Tcpu << endl << endl;
  cout<<"....................................................."<<endl<<endl;

  cout << "Speedup TCPU/TGPU_original= " << Tcpu / Tgpu_original << endl;
  cout<<"....................................................."<<endl<<endl;

  cout << "Speedup TCPU/TGPU_modificado= " << Tcpu / Tgpu_modificado << endl;
  cout<<"....................................................."<<endl<<endl;

  //**************************************************************************
  //***************************** REDUCCION CPU ******************************
  //**************************************************************************
  time=clock();

  long int suma_CPU = 0.0f;
  float media_CPU;
  for (int i = 0; i < nverts2; i++)
    suma_CPU += A[i];
  media_CPU = suma_CPU / nverts2;

  Tcpu_reduccion = (clock() - time) / CLOCKS_PER_SEC;

  cout << "Tiempo CPU algoritmo reduccion = " << Tcpu_reduccion << endl;
  cout << "Suma obtenida en el algoritmo de reduccion CPU = " << suma_CPU << endl;
  cout << "Media obtenida en el algoritmo de reduccion CPU = " << media_CPU << endl << endl;
  cout<<"....................................................."<<endl<<endl;

  cout << "Speedup TCPU_reduccion/TGPU_reduccion= " << Tcpu_reduccion / Tgpu_reduccion << endl;
  cout<<"....................................................."<<endl<<endl;


  bool errors=false;
  // Error Checking (CPU vs. GPU)
  for(int i = 0; i < nverts; i++)
    for(int j = 0; j < nverts; j++)
      if (abs(c_Out_M[i*nverts+j] - G.arista(i,j)) > 0){
        cout << "Error (" << i << "," << j << ")   " << c_Out_M[i*nverts+j] << "..." << G.arista(i,j) << endl;
        errors=true;
      }


  if (!errors){
    cout<<"....................................................."<<endl;
    cout<< "WELL DONE!!! No errors found ............................"<<endl;
    cout<<"....................................................."<<endl<<endl;
  }

  hipFree(d_In_M);
}
